#include "hip/hip_runtime.h"

#include "cudaAcceleration.h"

#ifdef USE_CUDA

#include "cudaAcc_data.h"
#include "cudaAcc_utilities.h"

//#define B 2


#define RPI 64
#define RPIB 8
#define RPBY 4

#define B 8
#define RPS 64
//must be 256 in current implementation
#define RDP 256

float2 *dev_AutoCorrIn;
float2 *dev_AutoCorrOut;

bool gCudaAutocorrelation = false;

float3 *blockSums;
float3 *dev_ac_partials;

//float ac_TotalSum;
//float ac_Peak;
//int ac_PeakBin;


__global__ void 
//__launch_bounds__(RPI, 8)
ac_RepackInputKernelP(float *PowerSpectrum, float2 *dct_In, int acfftlen) 
{
  int sidx = (threadIdx.x + blockIdx.x*RPI + (blockIdx.y*RPI*RPIB)); 
  int nDestPoints = acfftlen * 4; // (gridDim.y*RPI*RPIB)*4;
  int didx1 = sidx<<4; 
  int didx2 = ((nDestPoints-2)<<3)-didx1; 

  PowerSpectrum += sidx + blockIdx.z * acfftlen;
  float4 *dct1 = (float4*)((char *)dct_In + didx1 + blockIdx.z * acfftlen * 32);
  float4 *dct2 = (float4*)((char *)dct_In + didx2 + blockIdx.z * acfftlen * 32);
  float4 t = make_float4(0.0f, 0.0f, PowerSpectrum[0], 0.0f);
      
  *dct1 = t;
  *dct2 = t;
}



__global__ void 
//__launch_bounds__(RPS, 8)
ac_RepackScaleKernelP(float2 *src, float2 *dst, int acfftlen) 
{
  int didx = ((threadIdx.x + blockIdx.x*RPS + blockIdx.y*RPS*B));  //packing into float2s
  int sidx = didx << 1; //((threadIdx.x + blockIdx.x*RPS*B)*2);

//printf("bx=%d, by=%d, tx=%d, sidx=%d\r\n", blockIdx.x, blockIdx.y, threadIdx.x, sidx);
  dst += didx + blockIdx.z * acfftlen;
  src += sidx + blockIdx.z * acfftlen * 4;

  float4 t = ((float4 *)src)[0];
  float2 a = make_float2(t.x, t.z);

  a.x *= a.x;
  a.y *= a.y;

  dst[0] = a;
}


extern  __shared__ float acpartial[]; 

__global__ void ac_reducePartial(float *ac, float3 *devpartials, int streamIdx)
{
  const int tid = threadIdx.x;
  const int idx = threadIdx.x + blockIdx.x*RDP;
  const int bid = blockIdx.x;
  int n = RDP>>1;
  
  float3 *acp = (float3 *)acpartial; 
  float tmp = ac[idx];
  acp[tid].z = idx;
  acp[tid].y = idx >= 1 ? tmp : 0.0f; 
  acp[tid].x = tmp;
  
  __syncthreads();
  
  volatile float3 *dp = &acp[tid];
  int fadd = n * 12;
#pragma unroll 2
  for(; n > 32; n >>= 1)
    {
      if(tid < n)
	{
	  float a =  __fadd_rn(dp[0].x, (*(float3*)(((char *)dp)+fadd)).x);
	  // peak power & its bin
	  float pp = (*(float3*)(((char *)dp)+fadd)).y;
	  float pb = (*(float3*)(((char *)dp)+fadd)).z;
	  fadd >>= 1;
	  bool b = pp > dp[0].y;
          dp[0].x = a;
	  if(b)
	    {
	      dp[0].y = pp;
	      dp[0].z = pb;
	    }
	}
      __syncthreads();
    }

#pragma unroll 6
  for(; n > 0; n >>= 1)
    {
      if(tid < n)
	{
	  float a =  __fadd_rn(dp[0].x, (*(float3*)(((char *)dp)+fadd)).x);
	  // peak power & its bin
	  float pp = (*(float3*)(((char *)dp)+fadd)).y;
	  float pb = (*(float3*)(((char *)dp)+fadd)).z;
	  fadd >>= 1;
	  bool b = pp > dp[0].y;
          dp[0].x = a;
	  if(b)
	    {
	      dp[0].y = pp;
	      dp[0].z = pb;
	    }
	}
    }

  if(tid == 0) 
    {        
      devpartials[bid] = make_float3(dp[0].x, dp[0].y, dp[0].z);
    }
}

__global__ void ac_reducePartial16(float *ac, float3 *devpartials, int ac_fftlen)
{
  const int tid = threadIdx.x;
  const int idx = threadIdx.x + blockIdx.x*RDP;
  const int bid = blockIdx.x;
  
  float3 *acp = (float3 *)acpartial; 
  for(int f = 0; f < 8; f++)
    {
      float tmp = ac[idx+f*ac_fftlen*2];
      acp[tid].z = idx;
      acp[tid].y = idx >= 1 ? tmp : 0.0f; 
      acp[tid].x = tmp;

      __syncthreads();
  
      volatile float3 *dp = &acp[tid];
      int n = RDP>>1;
      int fadd = n * 12;

#pragma unroll 2
      for(;n > 32; n >>= 1)
	{
	  if(tid < n)
	    {
	      float a =  __fadd_rn(dp[0].x, (*(float3*)(((char *)(dp))+fadd)).x);
	      // peak power & its bin
	      float pp = (*(float3*)(((char *)(dp))+fadd)).y;
	      float pb = (*(float3*)(((char *)(dp))+fadd)).z;
	      bool b = pp > dp[0].y;
	      dp[0].x = a;
	      if(b)
		{
		  dp[0].y = pp;
		  dp[0].z = pb;
		}
	    }
	  
	  fadd >>= 1;
	  __syncthreads();
	}
      
#pragma unroll 6
      for(; n > 0; n >>= 1)
	{
	  if(tid < n)
	    {
	      float a =  __fadd_rn(dp[0].x, (*(float3*)(((char *)(dp))+fadd)).x);
	      // peak power & its bin
	      float pp = (*(float3*)(((char *)(dp))+fadd)).y;
	      float pb = (*(float3*)(((char *)(dp))+fadd)).z;
	      bool b = pp > dp[0].y;
	      dp[0].x = a;
	      if(b)
		{
		  dp[0].y = pp;
		  dp[0].z = pb;
		}
	    }
	  fadd >>= 1;
	}
      
      
      if(tid == 0) 
	devpartials[bid + f*(ac_fftlen/2)/RDP] = make_float3(dp[0].x, dp[0].y, dp[0].z);
    }
}


//__device__ float2 testPtr[8][10];

int cudaAcc_FindAutoCorrelations(int ac_fftlen, int offset) 
{
  hipError_t err = hipStreamWaitEvent(cudaAutocorrStream, powerspectrumDoneEvent, 0);	

  dim3 block(RPI, 1, 1);
  dim3 grid(RPIB, (ac_fftlen + (block.x*RPIB) - 1) / (block.x*RPIB), 8); 
  dim3 block2(RPS, 1, 1);
  dim3 grid2(B, ((ac_fftlen>>1)+block2.x*B-1)/(block2.x*B), 8);

  //Jason: Use 4N-FFT method for Type 2 Discrete Cosine Tranform for now, to match fftw's REDFT10
  // 1 Autocorrelation from global powerspectrum at fft_num*ac_fft_len  (fft_num*ul_NumDataPoints )
  
  //Step 1: Preprocessing - repack relevant powerspectrum into a 4N array with 'real-even symmetry'
  CUDA_ACC_SAFE_LAUNCH( (ac_RepackInputKernelP<<<grid, block, 0, cudaAutocorrStream>>>(dev_PowerSpectrum + offset, dev_AutoCorrIn, ac_fftlen)),true);

  //Step 2: Process the 4N-FFT (Complex to Complex, size is 4 * ac_fft_len)
  hipfftExecC2C(cudaAutoCorr_plan, dev_AutoCorrIn, dev_AutoCorrOut, HIPFFT_FORWARD);

  //Step 3: Postprocess the FFT result (Scale, take powers & normalise), discarding unused data packing into AutoCorr_in first half for VRAM reuse
  //CUDA_ACC_SAFE_LAUNCH( (
  ac_RepackScaleKernelP<<<grid2, block2, 0, cudaAutocorrStream>>>( dev_AutoCorrOut, dev_AutoCorrIn, ac_fftlen);
  //),true);

  int len = ac_fftlen/2;
  int blksize = RDP; 
  dim3 block3(blksize, 1, 1);
  dim3 grid3(len/blksize, 1, 1);

  CUDA_ACC_SAFE_LAUNCH( (ac_reducePartial16<<<grid3, block3, 3072, cudaAutocorrStream>>>( (float *)(dev_AutoCorrIn), dev_ac_partials, ac_fftlen)),true); // dynamic shared size is len/RDP*sizeof(float3) -> limit 4608

/*
  int len = ac_fftlen/2;
  int blksize = RDP; 
  dim3 block3(blksize, 1, 1);
  dim3 grid3(len/blksize, 1, 1);

  for(int fft_num = 0; fft_num < 8; fft_num++)
    {
      CUDA_ACC_SAFE_LAUNCH( (ac_reducePartial<<<grid3, block3, 3072, cudaAutocorrStream>>>( (float *)(dev_AutoCorrIn + fft_num * ac_fftlen), dev_ac_partials+fft_num*(ac_fftlen/2)/RDP, fft_num)),true); // dynamic shared size is len/RDP*sizeof(float3) -> limit 4608
    }

*/

  err = hipMemcpyAsync(blockSums, dev_ac_partials, 8*(ac_fftlen/2)/RDP*sizeof(float3), hipMemcpyDeviceToHost, cudaAutocorrStream);
  if(hipSuccess != err) { fprintf(stderr, "Autocorr - memcpyAsync %d", 0); exit(0); }

  err = hipEventRecord(autocorrelationDoneEvent[0], cudaAutocorrStream);
  if(hipSuccess != err) { fprintf(stderr, "Autocorr done %d", 0); exit(0); }

  return 0;
}





// TODO (half done): start all autocorrs. start all datadownloads. do ALL peak finds. cudasync. process all autocorr results.

int cudaAcc_GetAutoCorrelation(float *AutoCorrelation, int ac_fftlen, int fft_num)
{
  int len = ac_fftlen/2;
  int blksize = RDP; 
  float rac_TotalSum = 0, ac_Peak = 0;
  int ac_PeakBin = 0;
  hipError_t err;

  if(fft_num == 0)
    {
      err = hipEventSynchronize(autocorrelationDoneEvent[0]); // host (CPU) code waits for the all (specific) GPU task to complete
      if(hipSuccess != err) { fprintf(stderr, "GetAutocorr - sync %d", fft_num); exit(0); }
    }

  for(int b = 0; b < len/blksize; b++)
    {
      rac_TotalSum += blockSums[b + fft_num*len/blksize].x;
      if(blockSums[b + fft_num*len/blksize].y > ac_Peak)
	{
	  ac_Peak = blockSums[b + fft_num*len/blksize].y;
	  ac_PeakBin = b;
	}
    }

  blockSums[fft_num*len/blksize].x = rac_TotalSum;
  blockSums[fft_num*len/blksize].y = ac_Peak;
  blockSums[fft_num*len/blksize].z = blockSums[fft_num*len/blksize + ac_PeakBin].z;
  
  return 0;
}


#endif //USE_CUDA
