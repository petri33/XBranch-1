#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2008 NVIDIA Corporation.  All rights reserved.
 *
 * NOTICE TO USER:   
 *
 * This source code is subject to NVIDIA ownership rights under U.S. and 
 * international Copyright laws.  
 *
 * NVIDIA MAKES NO REPRESENTATION ABOUT THE SUITABILITY OF THIS SOURCE 
 * CODE FOR ANY PURPOSE.  IT IS PROVIDED "AS IS" WITHOUT EXPRESS OR 
 * IMPLIED WARRANTY OF ANY KIND.  NVIDIA DISCLAIMS ALL WARRANTIES WITH 
 * REGARD TO THIS SOURCE CODE, INCLUDING ALL IMPLIED WARRANTIES OF 
 * MERCHANTABILITY, NONINFRINGEMENT, AND FITNESS FOR A PARTICULAR PURPOSE.   
 * IN NO EVENT SHALL NVIDIA BE LIABLE FOR ANY SPECIAL, INDIRECT, INCIDENTAL, 
 * OR CONSEQUENTIAL DAMAGES, OR ANY DAMAGES WHATSOEVER RESULTING FROM LOSS 
 * OF USE, DATA OR PROFITS, WHETHER IN AN ACTION OF CONTRACT, NEGLIGENCE 
 * OR OTHER TORTIOUS ACTION, ARISING OUT OF OR IN CONNECTION WITH THE USE 
 * OR PERFORMANCE OF THIS SOURCE CODE.  
 *
 * U.S. Government End Users.  This source code is a "commercial item" as 
 * that term is defined at 48 C.F.R. 2.101 (OCT 1995), consisting  of 
 * "commercial computer software" and "commercial computer software 
 * documentation" as such terms are used in 48 C.F.R. 12.212 (SEPT 1995) 
 * and is provided to the U.S. Government only as a commercial end item.  
 * Consistent with 48 C.F.R.12.212 and 48 C.F.R. 227.7202-1 through 
 * 227.7202-4 (JUNE 1995), all U.S. Government End Users acquire the 
 * source code with only those rights set forth herein.
 */

#include "cudaAcceleration.h"
#ifdef USE_CUDA

#include "cudaAcc_data.h"

#include "cudaAcc_utilities.h"

float3 *dev_PowerSpectrumSumMax;
float3 *PowerSpectrumSumMax;

extern __shared__ float3 partial[];
// At least 32elems per sum
template <int iterations>
__global__ void cudaAcc_summax32_kernel(float *input, float3 *output) 
{    
  const int tid = threadIdx.x;
  const int y = blockIdx.y * blockDim.y;
  const int gridX = blockDim.x;
  const int start = y * gridX * iterations;
  const int end = (y + 1) * gridX * iterations;
  

  float maximum = 0.0f;    
  int pos = 0;
  int i = start + tid;
  float *ip = &input[i];
  float val = *ip, val2, val3, val4;
  float sum = val, sum2 = 0.0f;

  if(tid > 0) 
    { // Original max omits first element of every fft
      if(val > maximum)
	pos = i;
      maximum = max(maximum, val);
    }

  ip = &input[start + tid + gridX];  
  for(i = start + tid + gridX; i < (end-gridX*3); i += gridX) 
    {
      val  = LDG_f_cs(ip, 0); ip += gridX;  
      val2 = LDG_f_cs(ip, 0); ip += gridX;  
      val3 = LDG_f_cs(ip, 0); ip += gridX;  
      val4 = LDG_f_cs(ip, 0); ip += gridX;  

      sum +=  val + val2;
      sum2 += val3 + val4;
      if(val > maximum)
	{
	  pos = i;
	  maximum = val;
	}
      i += gridX;
      
      if(val2 > maximum)
	{
	  pos = i;
	  maximum = val2;
	}
      i += gridX;
      
      if(val3 > maximum)
	{
	  pos = i;
          maximum = val3;
	}
      i += gridX;
      
      if(val4 > maximum)
	{
	  pos = i;
          maximum = val4;
	}
    }
  
  // tail
  for(; i < end; i += gridX) 
    {
      val = LDG_f_cs(ip, 0);
      ip += gridX;  
      sum = sum + val;
      if(val > maximum)
	{
	  pos = i;
	  maximum = val;
	}
    }

  float a, ay, az;
  float3 *pp = &partial[tid];
  *pp = make_float3(a = sum+sum2, ay = maximum, az = pos - start);
  int padd = (gridX>>1)*12;

  for(i = gridX >> 1; i > 0; i >>= 1) 
    {
      __syncthreads(); 
      if(tid < i) 
	{            
	  a = a + (*(float3 *)(((char *)pp)+padd)).x;
	  float by = (*(float3 *)(((char *)pp)+padd)).y;
	  float bz = (*(float3 *)(((char *)pp)+padd)).z;

	  if(by > ay)
	    {
	      ay = by;
	      az = bz;
	    }

	  pp[0].x = a;
	  pp[0].y = ay;
	  pp[0].z = az;
	  
	  padd >>= 1;
	}
    }

  if(tid == 0) 
    {        
      output[blockIdx.y] = make_float3(a, ay, az);
    }
}


//less than 32 elems
template<int n>
__global__ void cudaAcc_summax_kernel(float* input, float3* output)
{
  const int tid = threadIdx.x;
  const int y = blockIdx.y * blockDim.y;
  const int gridX = gridDim.x * blockDim.x;
  const int start = y * gridX;
  
  const int n1 = n - 1;
  const int width = blockDim.x / n;
  
  float val = input[start + tid];

  partial[tid] = make_float3(val, ((tid & n1) == 0 ? 0.0f : val), tid & n1); // Original max omits first element of every fft

  float a = partial[tid].x;
  float ay = partial[tid].y;
  float az = partial[tid].z;
  for(int i = n >> 1; i > 0; i >>= 1)
    {
      __syncthreads(); 
      if((tid & n1) < i) 
	{
	  a = a + partial[tid+i].x;
	  float by = partial[tid+i].y;
	  float bz = partial[tid+i].z;

	  if(by > ay)
	    {
	      ay = by;
	      az = bz;
	    }
	  
	  partial[tid].x = a;
	  partial[tid].y = ay;
	  partial[tid].z = az;
	}
    }
  __syncthreads();
  
  if(tid < width)
    {
      output[blockIdx.y * width + tid] = make_float3(a, ay, az); //partial[tid * n];
    }
}

//Jason:---------------------------------------------------------------------------
template<class T>
struct SharedMemory
{
  __device__ inline operator       T*()
  {
    extern __shared__ int __smem[];
    return (T*)__smem;
  }
  
  __device__ inline operator const T*() const
  {
    extern __shared__ int __smem[];
    return (T*)__smem;
  }
};

template <unsigned int fftlen> 
__global__ void cudaAcc_SM(float *PowerSpectrum, float3 *devPowerSpectrumSumMax) 
{
  int iblock = blockIdx.x + blockIdx.y * gridDim.x;
  int sidx = (iblock*blockDim.x + threadIdx.x); 
  int tid = threadIdx.x;
  
  float3 *sdata = SharedMemory<float3>();
  
  float mySum = sdata[tid].x = PowerSpectrum[sidx];
  sdata[tid].z = tid;
  sdata[tid].y = (tid == 0 ? 0.0f :  mySum);
  __syncthreads();

  if (tid < fftlen/2) // last part of reduction is warp synchronous;
    {
      volatile float3* smem = sdata;
      if(fftlen >=  64) 
	{
	  smem[tid].x = mySum = mySum + smem[tid + 32].x;
	  smem[tid].y = max(smem[tid].y, smem[tid + 32].y);
	  smem[tid].z = smem[tid].y > smem[tid + 32].y ? smem[tid].z : smem[tid + 32].z; 
	}
      if(fftlen >=  32) 
	{
	  smem[tid].x = mySum = mySum + smem[tid + 16].x;  
	  smem[tid].y = max(smem[tid].y, smem[tid + 16].y);
	  smem[tid].z = smem[tid].y > smem[tid + 16].y ? smem[tid].z : smem[tid + 16].z; 
	}
      if(fftlen >=  16) 
	{
	  smem[tid].x = mySum = mySum + smem[tid +  8].x;  
	  smem[tid].y = max(smem[tid].y, smem[tid + 8].y);
	  smem[tid].z = smem[tid].y > smem[tid + 8].y ? smem[tid].z : smem[tid + 8].z; 
	}
      if(fftlen >=   8) 
	{
	  smem[tid].x = mySum = mySum + smem[tid +  4].x;
	  smem[tid].y = max(smem[tid].y, smem[tid + 4].y);
	  smem[tid].z = smem[tid].y > smem[tid + 4].y ? smem[tid].z : smem[tid + 4].z; 
	}
      if(fftlen >=   4) 
	{
	  smem[tid].x = mySum = mySum + smem[tid +  2].x;  
	  smem[tid].y = max(smem[tid].y, smem[tid + 2].y);
	  smem[tid].z = smem[tid].y > smem[tid + 2].y ? smem[tid].z : smem[tid + 2].z; 
	}
      if(fftlen >=   2) 
	{
	  smem[tid].x = mySum = mySum + smem[tid +  1].x;  
	  smem[tid].y = max(smem[tid].y, smem[tid + 1].y);
	  smem[tid].z = smem[tid].y > smem[tid + 1].y ? smem[tid].z : smem[tid + 1].z; 
	}
    }
  
  if(tid==0)
    {
      devPowerSpectrumSumMax[sidx/fftlen] = sdata[0];
    }
}

// TODO: optimize Memcpy, download data only when are going to be reported: 
//							if (si.score > best_spike->score || best_spike->s.fft_len == 0)
//							if (si.s.peak_power > (swi.analysis_cfg.spike_thresh))
void cudaAcc_summax(int fftlen, int offset) 
{
  hipStreamWaitEvent(summaxStream, powerspectrumDoneEvent, 0);

  dim3 block2(fftlen, 1, 1);
  dim3 grid2 = grid2D((cudaAcc_NumDataPoints + block2.x - 1) / block2.x);
  
  if(fftlen >= 32 && cudaAcc_NumDataPoints/fftlen < 65536)
    {
      int optimal_block_x;
      //if(gCudaDevProps.major > 2)
	optimal_block_x = max(32, min(pow2((unsigned int) sqrt((float) (fftlen / 32)) * 32), 1024)); 
      //else
	//optimal_block_x = max(32, min(pow2((unsigned int) sqrt((float) (fftlen / 32)) * 32), 512));
      dim3 block(optimal_block_x, 1, 1);;
      dim3 grid(1, cudaAcc_NumDataPoints / fftlen, 1);     
      int iterations = fftlen/block.x;
      
    
      switch(iterations)
      {
        case 1:
	  CUDA_ACC_SAFE_LAUNCH((cudaAcc_summax32_kernel<1><<<grid, block, (block.x * sizeof(float3)),summaxStream>>>(dev_PowerSpectrum + offset, dev_PowerSpectrumSumMax)),true);
	  break;
        case 2:
	  CUDA_ACC_SAFE_LAUNCH((cudaAcc_summax32_kernel<2><<<grid, block, (block.x * sizeof(float3)),summaxStream>>>(dev_PowerSpectrum + offset, dev_PowerSpectrumSumMax)),true);
	  break;
        case 4:
	  CUDA_ACC_SAFE_LAUNCH((cudaAcc_summax32_kernel<4><<<grid, block, (block.x * sizeof(float3)),summaxStream>>>(dev_PowerSpectrum + offset, dev_PowerSpectrumSumMax)),true);
	  break;
        case 8:
	  CUDA_ACC_SAFE_LAUNCH((cudaAcc_summax32_kernel<8><<<grid, block, (block.x * sizeof(float3)),summaxStream>>>(dev_PowerSpectrum + offset, dev_PowerSpectrumSumMax)),true);
	  break;
        case 16:
	  CUDA_ACC_SAFE_LAUNCH((cudaAcc_summax32_kernel<16><<<grid, block, (block.x * sizeof(float3)),summaxStream>>>(dev_PowerSpectrum + offset, dev_PowerSpectrumSumMax)),true);
	  break;
        case 32:
	  CUDA_ACC_SAFE_LAUNCH((cudaAcc_summax32_kernel<32><<<grid, block, (block.x * sizeof(float3)),summaxStream>>>(dev_PowerSpectrum + offset, dev_PowerSpectrumSumMax)),true);
	  break;
        case 64:
	  CUDA_ACC_SAFE_LAUNCH((cudaAcc_summax32_kernel<64><<<grid, block, (block.x * sizeof(float3)),summaxStream>>>(dev_PowerSpectrum + offset, dev_PowerSpectrumSumMax)),true);
	  break;
        case 128:
	  CUDA_ACC_SAFE_LAUNCH((cudaAcc_summax32_kernel<128><<<grid, block, (block.x * sizeof(float3)),summaxStream>>>(dev_PowerSpectrum + offset, dev_PowerSpectrumSumMax)),true);
	  break;
        case 256:
	  CUDA_ACC_SAFE_LAUNCH((cudaAcc_summax32_kernel<256><<<grid, block, (block.x * sizeof(float3)),summaxStream>>>(dev_PowerSpectrum + offset, dev_PowerSpectrumSumMax)),true);
	  break;
	default:
	  fprintf(stderr, "cudaAcc_summax32_kernel template error\r\n");
	  fflush(stderr);
	  *(char *)0 = 0;
      }     
    }
  else 
    {
      // Occupancy Calculator: 128 for cc1.x, 256 for cc2.x
      dim3 block(128, 1, 1);
      if(gCudaDevProps.major >= 2) 
	{
#if CUDART_VERSION >= 3000
//	  hipFuncSetCacheConfig(reinterpret_cast<const void*>(cudaAcc_summax32_kernel), hipFuncCachePreferShared); // Set this at init time
#endif
	  block.x = 256; //256 //128
	}
      dim3 grid(1, cudaAcc_NumDataPoints / block.x, 1);
      switch (fftlen) 
	{
	case 128:
	  CUDA_ACC_SAFE_LAUNCH((cudaAcc_summax_kernel<128><<<grid, block, (block.x * sizeof(float3)),summaxStream>>>(dev_PowerSpectrum + offset, dev_PowerSpectrumSumMax)),true);
	  break;
	case 64:
	  CUDA_ACC_SAFE_LAUNCH((cudaAcc_summax_kernel<64><<<grid, block, (block.x * sizeof(float3)),summaxStream>>>(dev_PowerSpectrum + offset, dev_PowerSpectrumSumMax)),true);
	  break;
	case 32:
	  CUDA_ACC_SAFE_LAUNCH((cudaAcc_summax_kernel<32><<<grid, block, (block.x * sizeof(float3)),summaxStream>>>(dev_PowerSpectrum + offset, dev_PowerSpectrumSumMax)),true);
	  break;
	case 16:
	  CUDA_ACC_SAFE_LAUNCH((cudaAcc_summax_kernel<16><<<grid, block, (block.x * sizeof(float3)),summaxStream>>>(dev_PowerSpectrum + offset, dev_PowerSpectrumSumMax)),true);
	  break;
	case 8:
	  CUDA_ACC_SAFE_LAUNCH((cudaAcc_summax_kernel<8><<<grid, block, (block.x * sizeof(float3)),summaxStream>>>(dev_PowerSpectrum + offset, dev_PowerSpectrumSumMax)),true);
	  break;
	}
    }
  
  
  hipMemcpyAsync(PowerSpectrumSumMax, dev_PowerSpectrumSumMax, (cudaAcc_NumDataPoints / fftlen) * sizeof(*dev_PowerSpectrumSumMax), hipMemcpyDeviceToHost, summaxStream);    
  hipEventRecord(summaxDoneEvent, summaxStream);
}


void cudaAcc_summax_x(int fftlen) // not used anymore
{
  dim3 block2(fftlen, 1, 1);
  dim3 grid2 = grid2D((cudaAcc_NumDataPoints + block2.x - 1) / block2.x);
  
  if(fftlen >= 32 && cudaAcc_NumDataPoints/fftlen < 65536)
    {
      int optimal_block_x;

      optimal_block_x = max(32, min(pow2((unsigned int) sqrt((float) (fftlen / 32)) * 32), 1024));
      
      dim3 block(optimal_block_x, 1, 1);;
      dim3 grid(1, cudaAcc_NumDataPoints / fftlen, 1);     
      int iterations = fftlen/block.x;

      switch(iterations)
      {
        case 1:
	  CUDA_ACC_SAFE_LAUNCH((cudaAcc_summax32_kernel<1><<<grid, block, (block.x * sizeof(float3))>>>(dev_PowerSpectrum, dev_PowerSpectrumSumMax)),true);
	  break;
        case 2:
	  CUDA_ACC_SAFE_LAUNCH((cudaAcc_summax32_kernel<2><<<grid, block, (block.x * sizeof(float3))>>>(dev_PowerSpectrum, dev_PowerSpectrumSumMax)),true);
	  break;
        case 4:
	  CUDA_ACC_SAFE_LAUNCH((cudaAcc_summax32_kernel<4><<<grid, block, (block.x * sizeof(float3))>>>(dev_PowerSpectrum, dev_PowerSpectrumSumMax)),true);
	  break;
        case 8:
	  CUDA_ACC_SAFE_LAUNCH((cudaAcc_summax32_kernel<8><<<grid, block, (block.x * sizeof(float3))>>>(dev_PowerSpectrum, dev_PowerSpectrumSumMax)),true);
	  break;
        case 16:
	  CUDA_ACC_SAFE_LAUNCH((cudaAcc_summax32_kernel<16><<<grid, block, (block.x * sizeof(float3))>>>(dev_PowerSpectrum, dev_PowerSpectrumSumMax)),true);
	  break;
        case 32:
	  CUDA_ACC_SAFE_LAUNCH((cudaAcc_summax32_kernel<32><<<grid, block, (block.x * sizeof(float3))>>>(dev_PowerSpectrum, dev_PowerSpectrumSumMax)),true);
	  break;
        case 64:
	  CUDA_ACC_SAFE_LAUNCH((cudaAcc_summax32_kernel<64><<<grid, block, (block.x * sizeof(float3))>>>(dev_PowerSpectrum, dev_PowerSpectrumSumMax)),true);
	  break;
        case 128:
	  CUDA_ACC_SAFE_LAUNCH((cudaAcc_summax32_kernel<128><<<grid, block, (block.x * sizeof(float3))>>>(dev_PowerSpectrum, dev_PowerSpectrumSumMax)),true);
	  break;
        case 256:
	  CUDA_ACC_SAFE_LAUNCH((cudaAcc_summax32_kernel<256><<<grid, block, (block.x * sizeof(float3))>>>(dev_PowerSpectrum, dev_PowerSpectrumSumMax)),true);
	  break;
	default:
	  fprintf(stderr, "cudaAcc_summax32_kernel template error\r\n");
	  fflush(stderr);
	  *(char *)0 = 0;
      }     
    }
  else 
    {
      // Occupancy Calculator: 128 for cc1.x, 256 for cc2.x
      dim3 block(128, 1, 1);
      if(gCudaDevProps.major >= 2) 
	{
#if CUDART_VERSION >= 3000
//	  hipFuncSetCacheConfig(reinterpret_cast<const void*>(cudaAcc_summax32_kernel), hipFuncCachePreferShared); // Set this at init time
#endif
	  block.x = 256; //64
	}
      dim3 grid(1, cudaAcc_NumDataPoints / block.x, 1);
      switch (fftlen) 
	{
	case 128:
	  CUDA_ACC_SAFE_LAUNCH((cudaAcc_summax_kernel<128><<<grid, block, (block.x * sizeof(float3))>>>(dev_PowerSpectrum, dev_PowerSpectrumSumMax)),true);
	  break;
	case 64:
	  CUDA_ACC_SAFE_LAUNCH((cudaAcc_summax_kernel<64><<<grid, block, (block.x * sizeof(float3))>>>(dev_PowerSpectrum, dev_PowerSpectrumSumMax)),true);
	  break;
	case 32:
	  CUDA_ACC_SAFE_LAUNCH((cudaAcc_summax_kernel<32><<<grid, block, (block.x * sizeof(float3))>>>(dev_PowerSpectrum, dev_PowerSpectrumSumMax)),true);
	  break;
	case 16:
	  CUDA_ACC_SAFE_LAUNCH((cudaAcc_summax_kernel<16><<<grid, block, (block.x * sizeof(float3))>>>(dev_PowerSpectrum, dev_PowerSpectrumSumMax)),true);
	  break;
	case 8:
	  CUDA_ACC_SAFE_LAUNCH((cudaAcc_summax_kernel<8><<<grid, block, (block.x * sizeof(float3))>>>(dev_PowerSpectrum, dev_PowerSpectrumSumMax)),true);
	  break;
	}
    }
  
  
  hipMemcpyAsync(PowerSpectrumSumMax, dev_PowerSpectrumSumMax, (cudaAcc_NumDataPoints / fftlen) * sizeof(*dev_PowerSpectrumSumMax), hipMemcpyDeviceToHost);    
}

#endif //USE_CUDA
